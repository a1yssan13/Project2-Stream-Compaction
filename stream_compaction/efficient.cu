#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernScan(int n, int *data) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
            if (index >= n) {
                return; 
            }
            // up-sweep faze of array 
            for (int d = 0; d < ilog2ceil(n); d++) {
                int offset = 1 << d; // 2^d. offset used for child. 
                if ((index + 1) / offset % 2 == 0) {
                    data[index] = data[index - offset] + data[index]; 
                __syncthreads(); 
            }
            // down-sweep faze of array  
            // set root to zero. At each pass, a node passes its value to its left 
            // child, and sets the right child to left value + this node's value. 
            for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
                int offset = 1 << d;
                if ((index + 1) / offset % 2 == 0) {
                    int temp = data[index]; 
                    data[index] = data[index] + data[index - offset]; 
                    data[index - offset] = temp; 
                }
                __syncthreads(); 
            }
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int *dev_data; 
            dim3 blockSize = 1024;  
            dim3 numBlocks = (n + blockSize - 1) / blockSize;  

            hipMalloc(&dev_data, n * sizeof(int));
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

            kernScan<<<numBlocks, blockSize>>>(n, dev_data);

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_data);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            
            int *dev_bools, *dev_indices; 
            dim3 blockSize = 1024; 
            dim3 numBlocks = (n + blockSize - 1) / blockSize; 

            scan(n, odata, idata);

            hipMalloc(&dev_bools, n * sizeof(int));
            hipMalloc(&dev_indices, n * sizeof(int));
            hipMalloc(&dev_data, n * sizeof(int));
            hipMalloc(&dev_odata, n * sizeof(int));
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            
            kernMapToBoolean<<<numBlocks, blockSize>>>(n, dev_bools, dev_data);
            kernScatter<<<numBlocks, blockSize>>>(n, dev_odata, dev_data, dev_bools, dev_indices);

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_data);
            hipFree(dev_odata);
            
            timer().endGpuTimer();
            return -1;
        }
    }
}
